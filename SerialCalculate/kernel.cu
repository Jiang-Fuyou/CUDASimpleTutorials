#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

__global__ void SOS(int *result, int *input, const int num)
{
	int sum = 0;
	for (int i = 0; i < num; i++) {
		sum += input[i] * input[i];
	}
	
	*result = sum;
}

int main()
{
	const int n = 5;
	int h_a[n] = { 1, 2, 3, 4, 5 };
	int h_r;

	int *d_a, *d_r;
	hipMalloc((void **)&d_a, n * sizeof(int));
	hipMalloc((void **)&d_r, 1 * sizeof(int));

	hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);

	SOS<<<1, 1>>>(d_r, d_a, n);

	hipMemcpy(&h_r, d_r, 1 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_r);

	printf("Result: %d\n", h_r);
}
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define DATA_SIZE 1000

int data[DATA_SIZE];

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++) {
		number[i] = rand() % 10;
	}
}

//CUDA ��ʼ��
bool InitCUDA()
{
	int count;

	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;

	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

__global__ static void sumOfSquares(int *num, int *result, clock_t *time)
{
	int sum = 0;
	clock_t start = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
	*time = clock() - start;
}

int main()
{
	//CUDA ��ʼ��
	if (!InitCUDA()) {
		return 0;
	}

	//���������
	GenerateNumbers(data, DATA_SIZE);

	/*�����ݸ��Ƶ��Կ��ڴ���*/

	int *gpudata, *result;
	clock_t *time;

	//cudaMalloc ȡ��һ���Կ��ڴ� ( ����result�����洢������ )
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t));

	//cudaMemcpy ����������������Ƶ��Կ��ڴ��� 
	//cudaMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
	//cudaMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	// ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
	sumOfSquares <<<1, 1, 0 >>>(gpudata, result, time);


	/*�ѽ������ʾоƬ���ƻ����ڴ�*/

	int sum;
	clock_t time_used;

	//cudaMemcpy ��������Դ��и��ƻ��ڴ�
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

	//Free
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	printf("GPUsum: %d. Time: %d\n", sum, time_used);

	sum = 0;

	clock_t start = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i];
	}
	clock_t end = clock();

	printf("CPUsum: %d. Time: %d\n", sum, end - start);

	return 0;
}

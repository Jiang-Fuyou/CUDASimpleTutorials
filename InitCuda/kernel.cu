#include <stdio.h>

#include <hip/hip_runtime.h>

bool InitCuda()
{
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		printf("There is no device.\n");
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		printf("There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

int main()
{
	if (InitCuda()) 
		printf("CUDA initialized.\n");

	int i = 0;
	int n = 5;
	for (i = 0; i < n; i++) {}

	printf("i = %d, n = %d\n", i, n);

	return 0;
}